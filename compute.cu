#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "vector.h"
#include "config.h"

#define BLOCK_SIZE 256

vector3* nums;
vector3** accels;

__global__ void PCompute(vector3* nums, vector3** accels, vector3* d_vel, vector3* d_pos, double* d_mass) {
    int t = blockIdx.x * blockDim.x + threadIdx.x;
    int i = t / NUMENTITIES;
    int j = t % NUMENTITIES;

    accels[t] = &nums[t*NUMENTITIES];
    if (t < NUMENTITIES * NUMENTITIES) {
        if(i == j){
            FILL_VECTOR(accels[i][j], 0, 0, 0);
        }else{
            vector3 dist;

            //finding the distance of all 3 demimesnioal spaces
            dist[0] = d_pos[i][0] - d_pos[j][0];
            dist[1] = d_pos[i][1] - d_pos[j][1];
            dist[2] = d_pos[i][2] - d_pos[j][2];

            // calculating magnitude and acceleration 
            double mag_sq = dist[0] * dist[0] + dist[1] *dist[1] + dist[2] * dist[2];
            double mag = sqrt(mag_sq);
            double accelmag = -1 * GRAV_CONSTANT * d_mass[j]/mag_sq;
            FILL_VECTOR(accels[i][j], accelmag*dist[0]/mag, accelmag*dist[1]/mag, accelmag*dist[2]/mag);
        }
        
        vector3 accel_sum = {(double) *(accels[t])[0], (double) *(accels[t])[1], (double) *(accels[t])[2]};
        
        d_vel[i][0]+=accel_sum[0]*INTERVAL;
		d_pos[i][0]=d_vel[i][0]*INTERVAL;

		d_vel[i][1]+=accel_sum[1]*INTERVAL;
		d_pos[i][1]=d_vel[i][1]*INTERVAL;

		d_vel[i][2]+=accel_sum[2]*INTERVAL;
		d_pos[i][2]=d_vel[i][2]*INTERVAL;
    }
}

void compute() {
    vector3 *d_vel, *d_pos;
    double *d_mass;

    hipMallocManaged((void**) &d_vel, (sizeof(vector3) * NUMENTITIES));
    hipMallocManaged((void**) &d_pos, (sizeof(vector3) * NUMENTITIES));
	hipMallocManaged((void**) &d_mass, (sizeof(double) * NUMENTITIES));

    hipMemcpy(d_vel, hVel, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);
	hipMemcpy(d_pos, hPos, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);
	hipMemcpy(d_mass, mass, sizeof(double) * NUMENTITIES, hipMemcpyHostToDevice);

    hipMallocManaged((void**) &nums, sizeof(vector3)*NUMENTITIES*NUMENTITIES);
    hipMallocManaged((void**) &accels, sizeof(vector3*)*NUMENTITIES);

    int blockSize = 256; 
    int numBlocks = (NUMENTITIES + blockSize - 1) / blockSize;

    PCompute<<<numBlocks, blockSize>>>(nums, accels, d_vel, d_pos, d_mass);
    hipDeviceSynchronize();

    hipMemcpy(hVel, d_vel, sizeof(vector3) * NUMENTITIES, hipMemcpyDefault);
    hipMemcpy(hPos, d_pos, sizeof(vector3) * NUMENTITIES, hipMemcpyDefault);
    hipMemcpy(mass, d_mass, sizeof(double) * NUMENTITIES, hipMemcpyDefault);

    hipFree(accels);
    hipFree(nums);
}
